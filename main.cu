#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <string>
#include <fstream>
#include <algorithm> 

#ifdef __HIPCC__
//CUDA includes
#include <hip/hip_runtime.h>
#endif

// STIM includes
#include <stim/parser/arguments.h>
#include <stim/visualization/camera.h>
#include <stim/gl/gl_texture.h>
#include <stim/visualization/gl_network.h>
#include <stim/biomodels/network.h>
#include <stim/visualization/gl_aaboundingbox.h>
#include <stim/visualization/colormap.h>

// OpenGL includes
#include <GL/glut.h>
#include <GL/freeglut.h>


//********************parameter setting*******************
// visualization objects
stim::gl_aaboundingbox<float> bb;	// axis-aligned bounding box object
stim::camera cam;					// camera object

// overall parameters
unsigned num_nets = 0;					// number of networks that've been loaded
float sigma;							// resample rate
float threshold;						// metric acceptable value
float radius = 0.7;						// equals to radius
float delta;							// camera moving parameter
std::vector<float> colormap;			// random generated color set
stim::gl_texture<unsigned char> S;		// texture storing the image stack
float planes[3] = { 0.0f, 0.0f, 0.0f };	// plane position in world space
std::vector<std::string> main_menu_option = { "compare mode", "mapping mode", "volume display"};	// main menu options
std::vector<std::string> sub_menu_option = { "overlaid on", "overlaid off", "highlight on", "highlight off", "light on", "light off" };		// sub menu option
std::string menu_name;
GLuint cmap_tex = 0;					// texture name for the color map
std::vector<unsigned char> brewer_color_map;

// hard-coded parameters
float resample_rate = 0.5f;			// sample rate for the network (fraction of sigma used as the maximum sample rate)
float camera_factor = 1.2f;			// start point of the camera as a function of X and Y size
float orbit_factor = 0.01f;			// degrees per pixel used to orbit the camera
float zoom_factor = 10.0f;			// zooming factor
float radius_factor = 0.5f;			// radius changing factor

// networks
stim::gl_network<float> GT;			// ground truth network
stim::gl_network<float> T;			// test network
stim::gl_network<float> _GT;		// splitted GT
stim::gl_network<float> _T;			// splitted T
std::vector<unsigned> _gt_t;		// store indices of nearest edge in _T for _GT
std::vector<unsigned> _t_gt;		// store indices of nearest edge in _GT for _T

// flags
bool load_image_stack = false;			// flag indicates loading image stacks
bool render_overlaid_network = false;	// flag indicates render a transparant T overlaid on GT
bool open_light = false;				// flag indicates light on/off
bool highlight_difference = false;		// flag indicates highlight the difference between two networks
bool compare_mode = true;				// default mode is compare mode
bool mapping_mode = false;
bool volume_mode = false;

// glut event parameters
bool LButtonDown = false;			// true when left button down
bool RButtonDown = false;
int mouse_x;						// window x-coordinate
int mouse_y;						// window y-coordinate
int mods;							// special keyboard input


//********************parameter setting*******************
// set up the squash transform to whole screen
void glut_render_single_projection() {

	glMatrixMode(GL_PROJECTION);					// load the projection matrix for editing
	glLoadIdentity();								// start with the identity matrix
	int X = glutGet(GLUT_WINDOW_WIDTH);				// use the whole screen for rendering
	int Y = glutGet(GLUT_WINDOW_HEIGHT);
	glViewport(0, 0, X, Y);							// specify a viewport for the entire window
	float aspect = (float)X / (float)Y;				// calculate the aspect ratio
	gluPerspective(60, aspect, 0.1, 1000000);		// set up a perspective projection
}

// set up the squash transform to left half screen
void glut_render_left_projection() {

	glMatrixMode(GL_PROJECTION);					// load the projection matrix for editing
	glLoadIdentity();								// start with the identity matrix
	int X = glutGet(GLUT_WINDOW_WIDTH) / 2;			// only use half of the screen for the viewport
	int Y = glutGet(GLUT_WINDOW_HEIGHT);
	glViewport(0, 0, X, Y);							// specify the viewport on the left
	float aspect = (float)X / (float)Y;				// calculate the aspect ratio
	gluPerspective(60, aspect, 0.1, 1000000);		// set up a perspective projection
}

// set up the squash transform to right half screen
void glut_render_right_projection() {

	glMatrixMode(GL_PROJECTION);					// load the projection matrix for editing
	glLoadIdentity();								// start with the identity matrix
	int X = glutGet(GLUT_WINDOW_WIDTH) / 2;			// only use half of the screen for the viewport
	int Y = glutGet(GLUT_WINDOW_HEIGHT);
	glViewport(X, 0, X, Y);							// specify the viewport on the right
	float aspect = (float)X / (float)Y;				// calculate the aspect ratio
	gluPerspective(60, aspect, 0.1, 1000000);		// set up a perspective projection
}

// translate camera to origin
void glut_render_modelview() {

	glMatrixMode(GL_MODELVIEW);						// load the modelview matrix for editing
	glLoadIdentity();								// start with the identity matrix
	stim::vec3<float> eye = cam.getPosition();		// get the camera position (eye point)
	stim::vec3<float> focus = cam.getLookAt();		// get the camera focal point
	stim::vec3<float> up = cam.getUp();				// get the camera "up" orientation

	gluLookAt(eye[0], eye[1], eye[2], focus[0], focus[1], focus[2], up[0], up[1], up[2]);	// set up the OpenGL camera
}

// three axis slice
// draw x slice
void draw_x_slice(float p) {
	float x = p;
	float y = S.size(1);
	float z = S.size(2);

	float tx = p / S.size(0);		// normalization

	glBegin(GL_QUADS);
	glTexCoord3f(tx, 0, 0);
	glVertex3f(x, 0, 0);

	glTexCoord3f(tx, 0, 1);
	glVertex3f(x, 0, z);

	glTexCoord3f(tx, 1, 1);
	glVertex3f(x, y, z);

	glTexCoord3f(tx, 1, 0);
	glVertex3f(x, y, 0);
	glEnd();
}
// draw y slice
void draw_y_slice(float p) {
	float x = S.size(0);
	float y = p;
	float z = S.size(2);

	float ty = p / S.size(1);

	glBegin(GL_QUADS);
	glTexCoord3f(0, ty, 0);
	glVertex3f(0, y, 0);

	glTexCoord3f(0, ty, 1);
	glVertex3f(0, y, z);

	glTexCoord3f(1, ty, 1);
	glVertex3f(x, y, z);

	glTexCoord3f(1, ty, 0);
	glVertex3f(x, y, 0);
	glEnd();
}
// draw z slice
void draw_z_slice(float p) {
	float x = S.size(0);
	float y = S.size(1);
	float z = p;

	float tz = p / S.size(2);

	glBegin(GL_QUADS);
	glTexCoord3f(0, 0, tz);
	glVertex3f(0, 0, z);

	glTexCoord3f(0, 1, tz);
	glVertex3f(0, y, z);

	glTexCoord3f(1, 1, tz);
	glVertex3f(x, y, z);

	glTexCoord3f(1, 0, tz);
	glVertex3f(x, 0, z);
	glEnd();
}

// draw a bounding box around the data set
void draw_box() {
	float c[3] = { S.size(0), S.size(1), S.size(2) };
	glLineWidth(1.0);

	glBegin(GL_LINE_LOOP);
	glColor3f(0, 0, 0);
	glVertex3f(0, 0, 0);

	glColor3f(0, 1, 0);
	glVertex3f(0, c[1], 0);

	glColor3f(0, 1, 1);
	glVertex3f(0, c[1], c[2]);

	glColor3f(0, 0, 1);
	glVertex3f(0, 0, c[2]);
	glEnd();

	glBegin(GL_LINE_LOOP);
	glColor3f(1, 0, 0);
	glVertex3f(c[0], 0, 0);

	glColor3f(1, 1, 0);
	glVertex3f(c[0], c[1], 0);

	glColor3f(1, 1, 1);
	glVertex3f(c[0], c[1], c[2]);

	glColor3f(1, 0, 1);
	glVertex3f(c[0], 0, c[2]);
	glEnd();

	glBegin(GL_LINES);
	glColor3f(0, 0, 0);
	glVertex3f(0, 0, 0);
	glColor3f(1, 0, 0);
	glVertex3f(c[0], 0, 0);

	glColor3f(0, 1, 0);
	glVertex3f(0, c[1], 0);
	glColor3f(1, 1, 0);
	glVertex3f(c[0], c[1], 0);

	glColor3f(0, 1, 1);
	glVertex3f(0, c[1], c[2]);
	glColor3f(1, 1, 1);
	glVertex3f(c[0], c[1], c[2]);

	glColor3f(0, 0, 1);
	glVertex3f(0, 0, c[2]);
	glColor3f(1, 0, 1);
	glVertex3f(c[0], 0, c[2]);
	glEnd();
}

// draw the plane frame
void draw_frames() {
	float c[3] = { S.size(0), S.size(1), S.size(2) };			// store the size of the data set for all three dimensions

	glLineWidth(1.0);
	glColor3f(1, 0, 0);											// draw the X plane
	glBegin(GL_LINE_LOOP);
	glVertex3f(planes[0], 0, 0);
	glVertex3f(planes[0], c[1], 0);
	glVertex3f(planes[0], c[1], c[2]);
	glVertex3f(planes[0], 0, c[2]);
	glEnd();

	glColor3f(0, 1, 0);											// draw the Y plane
	glBegin(GL_LINE_LOOP);
	glVertex3f(0, planes[1], 0);
	glVertex3f(c[0], planes[1], 0);
	glVertex3f(c[0], planes[1], c[2]);
	glVertex3f(0, planes[1], c[2]);
	glEnd();

	glColor3f(0, 0, 1);											// draw the Z plane
	glBegin(GL_LINE_LOOP);
	glVertex3f(0, 0, planes[2]);
	glVertex3f(c[0], 0, planes[2]);
	glVertex3f(c[0], c[1], planes[2]);
	glVertex3f(0, c[1], planes[2]);
	glEnd();
}

// enforce bound
void enforce_bounds() {
	for (int d = 0; d < 3; d++) {
		if (planes[d] < 0) planes[d] = 0;
		if (planes[d] > S.size(d)) planes[d] = S.size(d);
	}
}

// glut light sourse
void glut_light() {
	stim::vec3<float> p1 = cam.getLookAt() + cam.getUp() * 100000;
	stim::vec3<float> p2 = cam.getPosition();

	// light source
	GLfloat global_ambient[] = { 0.4, 0.4, 0.4, 1.0 };
	GLfloat ambient[] = { 0.2, 0.2, 0.2, 1.0 };
	GLfloat diffuse1[] = { 1.0, 1.0, 1.0, 1.0 };
	GLfloat diffuse2[] = { 0.4, 0.4, 0.4, 1.0 };
	GLfloat specular[] = { 1.0, 1.0, 1.0, 1.0 };
	GLfloat position1[] = { p1[0], p1[1], p1[2], 1.0 };		// upper right light source
	GLfloat position2[] = { p2[0], p2[1], p2[2], 1.0 };		// lower left light source

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glShadeModel(GL_SMOOTH);

	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, global_ambient);

	glLightfv(GL_LIGHT0, GL_AMBIENT, ambient);				// set ambient for light 0
	glLightfv(GL_LIGHT0, GL_DIFFUSE, diffuse1);				// set diffuse for light 0
	glLightfv(GL_LIGHT0, GL_SPECULAR, specular);			// set specular for light 0
	glLightfv(GL_LIGHT0, GL_POSITION, position1);			// set position for light 0

	glLightfv(GL_LIGHT1, GL_AMBIENT, ambient);				// set ambient for light 1
	glLightfv(GL_LIGHT1, GL_DIFFUSE, diffuse2);				// set diffuse for light 1
	glLightfv(GL_LIGHT1, GL_SPECULAR, specular);			// set specular for light 1
	glLightfv(GL_LIGHT1, GL_POSITION, position2);			// set position for light 1
}

// dynamically set up menu
void glut_set_menu(int value) {

	// remove last time menu options then add new menu options
	switch (value) {
	case 1:
		for (int i = 0; i < 4; i++)
			glutRemoveMenuItem(1);
		if (open_light) {
			menu_name = main_menu_option[1];
			glutAddMenuEntry(menu_name.c_str(), 2);
			menu_name = sub_menu_option[5];
			glutAddMenuEntry(menu_name.c_str(), 5);
		}
		else {
			menu_name = sub_menu_option[0];
			glutAddMenuEntry(menu_name.c_str(), 6);
			for (int i = 1; i < 3; i++) {
				menu_name = main_menu_option[i];
				glutAddMenuEntry(menu_name.c_str(), i + 1);
			}
			menu_name = sub_menu_option[4];
			glutAddMenuEntry(menu_name.c_str(), 4);
		}
		break;
	case 2:
		for (int i = 0; i < 4; i++)
			glutRemoveMenuItem(1);
		if (open_light) {
			menu_name = main_menu_option[0];
			glutAddMenuEntry(menu_name.c_str(), 1);
			menu_name = sub_menu_option[2];
			glutAddMenuEntry(menu_name.c_str(), 8);
			menu_name = sub_menu_option[5];
			glutAddMenuEntry(menu_name.c_str(), 5);
		}
		else {
			menu_name = main_menu_option[0];
			glutAddMenuEntry(menu_name.c_str(), 1);
			menu_name = sub_menu_option[2];
			glutAddMenuEntry(menu_name.c_str(), 8);
			menu_name = main_menu_option[2];
			glutAddMenuEntry(menu_name.c_str(), 3);
			menu_name = sub_menu_option[4];
			glutAddMenuEntry(menu_name.c_str(), 4);
		}
		break;
	case 3:
		for (int i = 0; i < 4; i++)
			glutRemoveMenuItem(1);
		menu_name = main_menu_option[0];
		glutAddMenuEntry(menu_name.c_str(), 1);
		menu_name = main_menu_option[1];
		glutAddMenuEntry(menu_name.c_str(), 2);
		break;
	case 4:
		for (int i = 0; i < 4; i++)
			glutRemoveMenuItem(1);
		if (mapping_mode) {
			if (highlight_difference) {
				menu_name = sub_menu_option[3];
				glutAddMenuEntry(menu_name.c_str(), 9);
			}
			else {
				menu_name = main_menu_option[0];
				glutAddMenuEntry(menu_name.c_str(), 1);
				menu_name = sub_menu_option[2];
				glutAddMenuEntry(menu_name.c_str(), 8);
			}
		}
		else {
			menu_name = main_menu_option[1];
			glutAddMenuEntry(menu_name.c_str(), 2);
		}
		menu_name = sub_menu_option[5];
		glutAddMenuEntry(menu_name.c_str(), 5);
		break;
	case 5:
		for (int i = 0; i < 4; i++)
			glutRemoveMenuItem(1);
		if (compare_mode) {
			menu_name = sub_menu_option[0];
			glutAddMenuEntry(menu_name.c_str(), 6);
			for (int i = 1; i < 3; i++) {
				menu_name = main_menu_option[i];
				glutAddMenuEntry(menu_name.c_str(), i + 1);
			}
		}
		if (mapping_mode) {
			if (highlight_difference) {
				menu_name = sub_menu_option[3];
				glutAddMenuEntry(menu_name.c_str(), 9);
			}
			else {
				menu_name = main_menu_option[0];
				glutAddMenuEntry(menu_name.c_str(), 1);
				menu_name = sub_menu_option[2];
				glutAddMenuEntry(menu_name.c_str(), 8);
				menu_name = main_menu_option[2];
				glutAddMenuEntry(menu_name.c_str(), 3);
			}
		}
		menu_name = sub_menu_option[4];
		glutAddMenuEntry(menu_name.c_str(), 4);
		break;
	case 6:
		for (int i = 0; i < 3; i++)
			glutRemoveMenuItem(2);
		menu_name = sub_menu_option[1];
		glutChangeToMenuEntry(1, menu_name.c_str(), 7);
		break;
	case 7:
		menu_name = sub_menu_option[0];
		glutChangeToMenuEntry(1, menu_name.c_str(), 6);
		for (int i = 1; i < 3; i++) {
			menu_name = main_menu_option[i];
			glutAddMenuEntry(menu_name.c_str(), i + 1);
		}
		if (open_light) {
			menu_name = sub_menu_option[5];
			glutAddMenuEntry(menu_name.c_str(), 5);
		}
		else {
			menu_name = sub_menu_option[4];
			glutAddMenuEntry(menu_name.c_str(), 4);
		}
		break;
	case 8:
		for (int i = 0; i < 4; i++)
			glutRemoveMenuItem(1);
		if (open_light) {
			menu_name = sub_menu_option[3];
			glutAddMenuEntry(menu_name.c_str(), 9);
			menu_name = sub_menu_option[5];
			glutAddMenuEntry(menu_name.c_str(), 5);
		}
		else {
			menu_name = sub_menu_option[3];
			glutAddMenuEntry(menu_name.c_str(), 9);
			menu_name = sub_menu_option[4];
			glutAddMenuEntry(menu_name.c_str(), 4);
		}
		break;
	case 9:
		for (int i = 0; i < 4; i++)
			glutRemoveMenuItem(1);
		if (open_light) {
			menu_name = main_menu_option[0];
			glutAddMenuEntry(menu_name.c_str(), 1);
			menu_name = sub_menu_option[2];
			glutAddMenuEntry(menu_name.c_str(), 8);
			menu_name = sub_menu_option[5];
			glutAddMenuEntry(menu_name.c_str(), 5);
		}
		else {
			menu_name = main_menu_option[0];
			glutAddMenuEntry(menu_name.c_str(), 1);
			menu_name = sub_menu_option[2];
			glutAddMenuEntry(menu_name.c_str(), 8);
			menu_name = main_menu_option[2];
			glutAddMenuEntry(menu_name.c_str(), 3);
			menu_name = sub_menu_option[4];
			glutAddMenuEntry(menu_name.c_str(), 4);
		}
		break;
	}
}

// defines camera motion based on mouse dragging
void glut_motion(int x, int y) {

	int mods = glutGetModifiers();
	if (LButtonDown == true && RButtonDown == false && mods == 0) {

		float theta = orbit_factor * (mouse_x - x);		// determine the number of degrees along the x-axis to rotate
		float phi = orbit_factor * (y - mouse_y);		// number of degrees along the y-axis to rotate

		cam.OrbitFocus(theta, phi);						// rotate the camera around the focal point
	}
	else if (mods != 0) {
		float dx = (float)(x - mouse_x);
		float dist = dx;								// calculate the distance that the mouse moved in pixel coordinates
		float sdist = dist;								// scale the distance by the sensitivity
		if (mods == GLUT_ACTIVE_SHIFT) {				// if the SHIFT key is pressed
			planes[0] += (sdist)* S.spacing(0);			// move the X plane based on the mouse wheel direction
		}
		else if (mods == GLUT_ACTIVE_CTRL) {			// if the CTRL key is pressed
			planes[1] += (sdist)* S.spacing(1);			// move the Y plane based on the mouse wheel direction
		}
		else if (mods == GLUT_ACTIVE_ALT) {				// if hte ALT key is pressed
			planes[2] += (sdist)* S.spacing(2);			// move the Z plane based on the mouse wheel direction
		}
		enforce_bounds();
	}

	mouse_x = x;										// update the mouse position
	mouse_y = y;

	glutPostRedisplay();								// re-draw the visualization
}

// sets the menu options
void glut_menu(int value) {

	if (value == 1) {									// menu 1 represents comparing mode
		compare_mode = true;
		mapping_mode = false;
		volume_mode = false;
	}
	if (value == 2) {									// menu 2 represents mapping mode
		compare_mode = false;
		mapping_mode = true;
		volume_mode = false;
	}
	if (value == 3) {									// menu 3 represents volume mode
		compare_mode = false;
		mapping_mode = false;
		volume_mode = true;
	}
	if (value == 4) {									// menu 4 represents open light
		open_light = true;
		glEnable(GL_LIGHTING);
		glEnable(GL_LIGHT0);
		glEnable(GL_LIGHT1);
	}
	if (value == 5) {									// menu 5 represents close light
		open_light = false;
		glDisable(GL_LIGHTING);
		glDisable(GL_LIGHT0);
		glDisable(GL_LIGHT1);
	}
	if (value == 6)										// menu 6 represents render overlaid network
		render_overlaid_network = true;
	if (value == 7)										// menu 7 represents erase overlaid network
		render_overlaid_network = false;
	if (value == 8)										// menu 8 represents turn on highlight
		highlight_difference = true;
	if (value == 9)										// menu 9 represents turn off highlight
		highlight_difference = false;

	glut_set_menu(value);

	glutPostRedisplay();
}

// get click window coordinates
void glut_mouse(int button, int state, int x, int y) {

	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN) {
		mouse_x = x;
		mouse_y = y;
		LButtonDown = true;
	}
	else if (button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN) {
		mouse_x = x;
		mouse_y = y;
		RButtonDown = true;
	}
	else if (button == GLUT_LEFT_BUTTON && state == GLUT_UP) {
		mouse_x = x;
		mouse_y = y;
		LButtonDown = false;
	}
	else if (button == GLUT_RIGHT_BUTTON && state == GLUT_UP) {
		mouse_x = x;
		mouse_y = y;
		RButtonDown = false;
	}
}

// define camera move based on mouse wheel move
void glut_wheel(int wheel, int direction, int x, int y) {

	int mods = glutGetModifiers();
	if (mods == GLUT_ACTIVE_SHIFT) {					// if the SHIFT key is pressed
		planes[0] += (direction)* S.spacing(0);			// move the X plane based on the mouse wheel direction
	}
	else if (mods == GLUT_ACTIVE_CTRL) {				// if the CTRL key is pressed
		planes[1] += (direction)* S.spacing(1);			// move the Y plane based on the mouse wheel direction
	}
	else if (mods == GLUT_ACTIVE_ALT) {					// if hte ALT key is pressed
		planes[2] += (direction)* S.spacing(2);			// move the Z plane based on the mouse wheel direction
	}
	else {
		if (direction > 0)								// if it is button 3(up), move closer
			delta = zoom_factor;
		else											// if it is button 4(down), leave farther
			delta = -zoom_factor;
	}
	cam.Push(delta);
	enforce_bounds();
	glutPostRedisplay();
}

// define keyboard inputs
void glut_keyboard(unsigned char key, int x, int y) {

	// register different keyboard operation
	switch (key) {

	// zooming
	case 'w':						// if keyboard 'w' is pressed, then move closer
		delta = zoom_factor;
		cam.Push(delta);
		break;
	case 's':						// if keyboard 's' is pressed, then leave farther
		delta = -zoom_factor;
		cam.Push(delta);
		break;

	// resample and re-render the cylinder in different radius
	case 'd':						// if keyboard 'd' is pressed, then increase radius by radius_factor
		radius += radius_factor;
		break;
	case 'a':						// if keyboard 'a' is pressed, then decrease radius by radius_factor
		radius -= radius_factor;
		// get rid of the degenerated case when radius decrease below 0
		if (radius < 0.001f)
			radius = 0.2;
		break;

	// close window and exit application
	case 27:						// if keyboard 'ESC' is pressed, then exit
		exit(0);
	}
	glutPostRedisplay();
}

// main render function
void glut_render() {
	
	glut_light();												// set up light
	
	if (num_nets == 1) {										// if a single network is loaded
		glEnable(GL_DEPTH_TEST);								// enable depth
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);		// clear the screen
		glut_render_single_projection();						// fill the entire viewport
		glut_render_modelview();								// set up the modelview matrix with camera details
		if (volume_mode) {
			draw_box();
			draw_frames();
			glEnable(GL_TEXTURE_3D);							// enable 3D texture mapping
			S.bind();											// bind the texture
			draw_x_slice(planes[0]);							// draw the X plane
			draw_y_slice(planes[1]);							// draw the Y plane
			draw_z_slice(planes[2]);							// draw the Z plane
			glDisable(GL_TEXTURE_3D);							// disable 3D texture mapping
		}
		GT.glCenterline0();										// render the GT network (the only one loaded)
		glDisable(GL_DEPTH_TEST);
	}

	else if (num_nets == 2) {									// if two networks are loaded
		glEnable(GL_DEPTH_TEST);								// enable depth
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);		// clear the screen
		
		// left projection
		glut_render_left_projection();							// set up a projection for the left half of the window
		glut_render_modelview();								// set up the modelview matrix using camera details
		if (compare_mode) {										// compare mode
			glEnable(GL_TEXTURE_1D);							// enable texture mapping
			if (!open_light)
				glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);	// texture map will be used as the network color
			else
				glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_MODULATE);// map light to texture

			_GT.glCylinder(sigma, radius);						// render the GT network
			if (render_overlaid_network) {
				glDisable(GL_TEXTURE_1D);							// temporarily disable texture
				glEnable(GL_BLEND);									// enable color blend
				glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);	// set blend function
				glDisable(GL_DEPTH_TEST);							// should disable depth
				glColor4f(0.8f, 0.8f, 0.8f, 0.2f);
				_T.glAdjointCylinder(sigma, radius);
				glDisable(GL_BLEND);
				glEnable(GL_DEPTH_TEST);
				glEnable(GL_TEXTURE_1D);							// re-enable texture
				glColor4f(1.0f, 1.0f, 1.0f, 1.0f);
			}
		}
		else if (mapping_mode) {
			glEnable(GL_COLOR_MATERIAL);
			if (!highlight_difference)
				_GT.glRandColorCylinder(0, _gt_t, colormap, sigma, radius);
			else
				_GT.glDifferenceCylinder(0, _gt_t, colormap, sigma, radius);
		}
		else if (volume_mode) {
			draw_box();
			draw_frames();
			glEnable(GL_TEXTURE_3D);								// enable 3D texture mapping
			S.bind();												// bind the texture
			draw_x_slice(planes[0]);								// draw the X plane
			draw_y_slice(planes[1]);								// draw the Y plane
			draw_z_slice(planes[2]);								// draw the Z plane
			glDisable(GL_TEXTURE_3D);								// disable 3D texture mapping
			_GT.glCylinder(sigma, radius);
		}

		// right projection
		glut_render_right_projection();							// set up a projection for the right half of the window
		glut_render_modelview();								// set up the modelview matrix using camera details
		if (compare_mode) {										// compare mode
			_T.glCylinder(sigma, radius);						// render the GT network
			glDisable(GL_TEXTURE_1D);
			if (render_overlaid_network) {
				glEnable(GL_BLEND);									// enable color blend
				glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);	// set blend function
				glDisable(GL_DEPTH_TEST);							// should disable depth
				glColor4f(0.8f, 0.8f, 0.8f, 0.2f);
				_GT.glAdjointCylinder(sigma, radius);
				glDisable(GL_BLEND);
				glColor4f(1.0f, 1.0f, 1.0f, 1.0f);
			}
		}
		else if (mapping_mode) {
			if (!highlight_difference)
				_T.glRandColorCylinder(1, _t_gt, colormap, sigma, radius);
			else
				_T.glDifferenceCylinder(1, _t_gt, colormap, sigma, radius);
			glDisable(GL_COLOR_MATERIAL);
		}
		else if (volume_mode) {
			draw_box();
			draw_frames();
			glEnable(GL_TEXTURE_3D);								// enable 3D texture mapping
			S.bind();												// bind the texture
			draw_x_slice(planes[0]);								// draw the X plane
			draw_y_slice(planes[1]);								// draw the Y plane
			draw_z_slice(planes[2]);								// draw the Z plane
			glDisable(GL_TEXTURE_3D);								// disable 3D texture mapping
			_T.glCylinder(sigma, radius);
		}
		sigma = radius;
	}

	if (num_nets == 2) {												// works only with two networks
		std::ostringstream ss;
		if (mapping_mode)												// if it is in mapping mode
			ss << "Mapping Mode";
		else if (compare_mode)
			ss << "Compare Mode";										// default mode is compare mode
		else
			ss << "Volume Display";

		if (open_light)
			glDisable(GL_LIGHTING);
		glMatrixMode(GL_PROJECTION);									// set up the 2d viewport for mode text printing
		glPushMatrix();
		glLoadIdentity();
		int X = glutGet(GLUT_WINDOW_WIDTH);								// get the current window width
		int Y = glutGet(GLUT_WINDOW_HEIGHT);							// get the current window height
		glViewport(0, 0, X / 2, Y);										// locate to left bottom corner
		gluOrtho2D(0, X, 0, Y);											// define othogonal aspect
		glColor3f(0.8f, 0.0f, 0.0f);									// using red to show mode

		glMatrixMode(GL_MODELVIEW);
		glPushMatrix();
		glLoadIdentity();

		glRasterPos2f(0, 5);											//print text in the left bottom corner
		glutBitmapString(GLUT_BITMAP_TIMES_ROMAN_24, (const unsigned char*)(ss.str().c_str()));

		glPopMatrix();
		glMatrixMode(GL_PROJECTION);
		glPopMatrix();
		glColor3f(1.0f, 1.0f, 1.0f);									//clear red color
		if (open_light)
			glEnable(GL_LIGHTING);
	}

	glutSwapBuffers();
}

#define BREWER_CTRL_PTS 11										// number of control points in the Brewer map
void texture_initialize() {

	//define the colormap
	static float  brewer_map[BREWER_CTRL_PTS][3] = {			// generate a Brewer color map (blue to red)
		{ 0.192157f, 0.211765f, 0.584314f },
		{ 0.270588f, 0.458824f, 0.705882f },
		{ 0.454902f, 0.678431f, 0.819608f },
		{ 0.670588f, 0.85098f, 0.913725f },
		{ 0.878431f, 0.952941f, 0.972549f },
		{ 1.0f, 1.0f, 0.74902f },
		{ 0.996078f, 0.878431f, 0.564706f },
		{ 0.992157f, 0.682353f, 0.380392f },
		{ 0.956863f, 0.427451f, 0.262745f },
		{ 0.843137f, 0.188235f, 0.152941f },
		{ 0.647059f, 0.0f, 0.14902f }
	};

	glGenTextures(1, &cmap_tex);								// generate a texture map name
	glBindTexture(GL_TEXTURE_1D, cmap_tex);						// bind the texture map

	glTexParameteri(GL_TEXTURE_1D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);		// enable linear interpolation
	glTexParameteri(GL_TEXTURE_1D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_1D, GL_TEXTURE_WRAP_S, GL_CLAMP);			// clamp the values at the minimum and maximum
	glTexImage1D(GL_TEXTURE_1D, 0, 3, BREWER_CTRL_PTS, 0, GL_RGB, GL_FLOAT,	// upload the texture map to the GPU
		brewer_map);
	if (load_image_stack == 1) {
		S.attach();												// attach 3D texture
	}
}

// initialize the OpenGL (GLUT) window, including starting resolution, callbacks, texture maps, and camera
void glut_initialize() {

	int myargc = 1;												// GLUT requires arguments, so create some bogus ones
	char* myargv[1];
	myargv[0] = strdup("netmets");

	glutInit(&myargc, myargv);									// pass bogus arguments to glutInit()
	glutSetOption(GLUT_MULTISAMPLE, 8);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);	// generate a color buffer, depth buffer, and enable double buffering
	glutInitWindowPosition(100, 100);							// set the initial window position
	glutInitWindowSize(320, 320);								// set the initial window size
	glutCreateWindow("NetMets - STIM Lab, UH");					// set the dialog box title

#ifdef _WIN32
	GLenum err = glewInit();									// initialize GLEW (necessary for Windows)
	if (GLEW_OK != err) {										// eror with GLEW
		std::cout << "Error with GLEW: " << glewGetErrorString(err) << std::endl;
		exit(1);
	}
#endif

	// register callback functions
	glutDisplayFunc(glut_render);				// function executed for rendering - renders networks
	glutMouseFunc(glut_mouse);					// executed on a mouse click - sets starting mouse positions for rotations
	glutMotionFunc(glut_motion);				// executed when the mouse is moved while a button is pressed
	glutKeyboardFunc(glut_keyboard);			// register keyboard callback
	glutMouseWheelFunc(glut_wheel);				// register mouse wheel callback

	// set up initial menu
	glutCreateMenu(glut_menu);					// register menu option callback
	for (int i = 0; i < 3; i++) {
		menu_name = main_menu_option[i];
		glutAddMenuEntry(menu_name.c_str(), i + 1);
	}
	menu_name = sub_menu_option[4];
	glutAddMenuEntry(menu_name.c_str(), 5);
	menu_name = sub_menu_option[0];
	glutChangeToMenuEntry(1, menu_name.c_str(), 6);
	glutAttachMenu(GLUT_RIGHT_BUTTON);			// register right mouse to open menu option

	texture_initialize();						// set up texture mapping (create texture maps, enable features)

	stim::vec3<float> c = bb.center();			// get the center of the network bounding box
	// place the camera along the z-axis at a distance determined by the network size along x and y
	cam.setPosition(c + stim::vec<float>(0, 0, camera_factor * std::max(bb.size()[0], bb.size()[1])));
	cam.LookAt(c[0], c[1], c[2]);				// look at the center of the network
}


//********************segmentation fucntion********************
// set up device on gpu
#ifdef __HIPCC__
// set specific device to work on
void set_device(int &device) {
	int count;
	hipGetDeviceCount(&count);					// numbers of device that are available
	if (count < device + 1) {
		std::cout << "No such device available, please set another device" << std::endl;
		exit(1);
	}
}
#else
void set_device(int &device) {
	device = -1;								// set to default -1
}
#endif

// split and map two networks and fill the networks' R with metric information
void mapping(float sigma, int device, float threshold) {

	//GT = GT.compare(T, sigma, device);				// compare the ground truth to the test case - store errors in GT
	//T = T.compare(GT, sigma, device);				// compare the test case to the ground truth - store errors in T

	// compare and split two networks
	_GT.split(GT, T, sigma, device, threshold);
	_T.split(T, GT, sigma, device, threshold);

	// mapping two new splitted networks and get their edge relation
	_GT.mapping(_T, _gt_t, device, threshold);
	_T.mapping(_GT, _t_gt, device, threshold);

	// generate random color set based on the number of edges in GT
	size_t num = _gt_t.size();						// also create random color for unmapping edge, but won't be used though
	colormap.resize(3 * num);						// 3 portions compound RGB
	for (int i = 0; i < 3 * num; i++)
		colormap[i] = rand() / (float)RAND_MAX;		// set to [0, 1]

	float FPR = _GT.average();						// calculate the average metric
	float FNR = _T.average();

	std::cout << "FNR: " << FPR << std::endl;		// print false alarms and misses
	std::cout << "FPR: " << FNR << std::endl;
}

// writes features of the networks i.e average segment length, tortuosity, branching index, contraction, fractal dimension, number of end and branch points to a csv file
void features(std::string filename) {
	double avgL_t, avgL_gt, avgT_t, avgT_gt, avgB_t, avgB_gt, avgC_t, avgC_gt, avgFD_t, avgFD_gt;
	unsigned int e_t, e_gt, b_gt, b_t;
	avgL_gt = GT.Lengths();
	avgT_gt = GT.Tortuosities();
	avgL_t = T.Lengths();
	avgT_t = T.Tortuosities();
	avgB_gt = GT.BranchingIndex();
	avgB_t = T.BranchingIndex();
	avgC_gt = GT.Contractions();
	avgFD_gt = GT.FractalDimensions();
	avgC_t = T.Contractions();
	avgFD_t = T.FractalDimensions();
	e_gt = GT.EndP();
	e_t = T.EndP();
	b_gt = GT.BranchP();
	b_t = T.BranchP();
	std::ofstream myfile;
	myfile.open(filename.c_str());
	myfile << "Length, Tortuosity, Contraction, Fractal Dimension, Branch Points, End points, Branching Index, \n";
	myfile << avgL_gt << "," << avgT_gt << "," << avgC_gt << "," << avgFD_gt << "," << b_gt << "," << e_gt << "," << avgB_gt << std::endl;
	myfile << avgL_t << "," << avgT_t << "," << avgC_t << "," << avgFD_t << "," << b_t << "," << e_t << "," << avgB_t << std::endl;
	myfile.close();
}

// output an advertisement for the lab, authors, and usage information
void advertise() {
	std::cout << std::endl << std::endl;
	std::cout << "=========================================================================" << std::endl;
	std::cout << "Thank you for using the NetMets network comparison tool!" << std::endl;
	std::cout << "Scalable Tissue Imaging and Modeling (STIM) Lab, University of Houston" << std::endl;
	std::cout << "Developers: Jiaming Guo, David Mayerich" << std::endl;
	std::cout << "Source: https://git.stim.ee.uh.edu/segmentation/netmets" << std::endl;
	std::cout << "=========================================================================" << std::endl << std::endl;

	std::cout << "usage: netmets file1 file2 --sigma 3" << std::endl;
	std::cout << "            compare two .obj files with a tolerance of 3 (units defined by the network)" << std::endl << std::endl;
	std::cout << "       netmets file1 --gui" << std::endl;
	std::cout << "            load a file and display it using OpenGL" << std::endl << std::endl;
	std::cout << "       netmets file1 file2 --device 0" << std::endl;
	std::cout << "            compare two files using device 0 (if there isn't a gpu, use cpu)" << std::endl << std::endl;
	std::cout << "       netmets file1 file2 --mapping value" << std::endl;
	std::cout << "            mapping two files in random colors with a threshold of value" << std::endl << std::endl;
}

int main(int argc, char* argv[])
{
	stim::arglist args;						// create an instance of arglist

	// add arguments
	args.add("help", "prints this help");
	args.add("sigma", "force a sigma value to specify the tolerance of the network comparison", "3");
	args.add("gui", "display the network or network comparison using OpenGL");
	args.add("device", "choose specific device to run", "0");
	args.add("features", "save features to a CSV file, specify file name");
	args.add("threshold", "metric acceptable value", "0.6", "any real positive value");
	args.add("stack", "load the image stacks");
	args.add("spacing", "spacing between pixel samples in each dimension", "1.0 1.0 1.0", "any real positive value");

	args.parse(argc, argv);					// parse the user arguments

	if (args["help"].is_set()) {			// test for help
		advertise();						// output the advertisement
		std::cout << args.str();			// output arguments
		exit(1);							// exit
	}

	if (args.nargs() >= 1) {				// if at least one network file is specified
		num_nets = 1;						// set the number of networks to one
		std::vector<std::string> tmp = stim::parser::split(args.arg(0), '.');	// split the filename at '.'
		if ("swc" == tmp[1]) 				// loading swc file
			GT.load_swc(args.arg(0));		// load the specified file as the ground truth
		else if ("obj" == tmp[1])			// loading obj file
			GT.load_obj(args.arg(0));		// load the specified file as the ground truth
		else if ("nwt" == tmp[1])			// loading nwt file
			GT.loadNwt(args.arg(0));
		else {
			std::cout << "Invalid loading file" << std::endl;
			exit(1);
		}
	}

	if (args.nargs() == 2) {					// if two files are specified, they will be displayed in neighboring viewports and compared
		num_nets = 2;							// set the number of networks to two

		int device = args["device"].as_int();	// get the device value from the user
		set_device(device);

		sigma = args["sigma"].as_float();		// get the sigma value from the user

		if (args["features"].is_set())			// if the user wants to save features
			features(args["features"].as_string());

		threshold = args["threshold"].as_float();

		std::vector<std::string> tmp = stim::parser::split(args.arg(1), '.');	// split the filename at '.'
		if ("swc" == tmp[1]) 					// loading swc files
			T.load_swc(args.arg(1));            // load the second (test) network
		else if ("obj" == tmp[1])				// loading obj files
			T.load_obj(args.arg(1));
		else if ("nwt" == tmp[1])				// loading nwt file
			T.loadNwt(args.arg(1));
		else {
			std::cout << "Invalid loading file" << std::endl;
			exit(1);
		}

		GT = GT.resample(resample_rate * sigma);// resample both networks based on the sigma value
		T = T.resample(resample_rate * sigma);

		mapping(sigma, device, threshold);
	}

	// load image stack
	if (args["stack"].is_set()) {
		S.load_images(args["stack"].as_string());
		load_image_stack = true;
	}

	// set up spacing value, the vexel
	float sp[3] = { 1.0f, 1.0f, 1.0f };						// allocate variables for grid spacing
	if (args["spacing"].nargs() == 1)						// if only one argument is given
		sp[2] = (float)args["spacing"].as_float(0);			// assume that it's the z coordinate (most often anisotropic)
	else if (args["spacing"].nargs() == 3) {				// if three arguments are given
		sp[0] = (float)args["spacing"].as_float(0);			// set the arguments as expected
		sp[1] = (float)args["spacing"].as_float(1);
		sp[2] = (float)args["spacing"].as_float(2);
	}
	S.spacing(sp[0], sp[1], sp[2]);							// set the spacing between samples

	// set start plane at one quater
	planes[0] = S.size(0) / 4.0f;							// initialize the start positions for the orthogonal display planes
	planes[1] = S.size(1) / 4.0f;
	planes[2] = S.size(2) / 4.0f;

	//if a GUI is requested, display the network using OpenGL
	if (args["gui"].is_set()) {

		bb = GT.boundingbox();					// generate a bounding volume		
		glut_initialize();						// create the GLUT window and set callback functions		
		glutMainLoop();							// enter GLUT event processing cycle
	}
}